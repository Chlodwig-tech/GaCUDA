#include "hip/hip_runtime.h"
#ifndef SPOPULATION_CU
#define SPOPULATION_CU

#include "kernels.cu"

#define CUDA_CALL(x, message) {if((x) != hipSuccess) { \
    printf("Error - %s(%d)[%s]: %s\n", __FILE__, __LINE__, message, hipGetErrorString(x)); \
    exit(EXIT_FAILURE); }}

enum MUTATION{ // mutations
    MUTATION_INVERSION,
    MUTATION_OWN, 
    MUTATION_SCRAMBLE,
    MUTATION_SWAP
};

enum CROSSOVER{ // crossovers
    CROSSOVER_ARITHMETIC,
    CROSSOVER_OWN,
    CROSSOVER_SINGLE_POINT,
    CROSSOVER_TWO_POINT,
    CROSSOVER_UNIFORM,
};

template<typename T> class Population{
protected:
    using DNA = typename T::DNA_t;
    using Tfitness = typename T::Tfitness_t;

    T *organisms;
    T *children;
    T* *porganisms;
    T* *pchildren;
    T helper;
    bool *ichildren;
    int size;
    hipStream_t stream;

public:
    Population(int size);
    ~Population();
    template<typename r> void random(r a, r b);
    template<typename r> void brandom(r a, r b, int nthreads=1024);
    template<typename r> void linspace(r a, r b, bool endpoint=true);
    template<typename r> void blinspace(r a, r b, bool endpoint=true);
    template<typename r> void logspace(r a, r b, DNA base, bool endpoint=true);
    template<typename r> void blogspace(r a, r b, DNA base, bool endpoint=true);
    template<typename r> void plinspace(r a, r b, bool endpoint=true);
    template<typename r> void bplinspace(r a, r b, bool endpoint=true);
    template<typename r> void plogspace(r a, r b, DNA base, bool endpoint=true);
    template<typename r> void bplogspace(r a, r b, DNA base, bool endpoint=true);
    void fitness();
    void bfitness(int nthreads=1024);
    void mutate(MUTATION mutation_type, float probability=1.0f);
    void crossover(CROSSOVER crossover_type, float probability=1.0f);
    void sortAll();
    void sortOrganisms();
    void init_organisms();
    void init_organisms_with_tid();
    void init_organisms_with_val(DNA val);
    void binit_organisms_with_val(DNA val);

    void printP(int max=-1);
    void print(int max=-1);
    void print_childrenP(int max=-1);
    void print_children(int max=-1);

    hipStream_t* getStream();
};

template<typename T> Population<T>::Population(int size) : size(size){
    CUDA_CALL(hipMalloc((void **)&organisms, size * sizeof(T)), "Population organisms hipMalloc");
    CUDA_CALL(hipMalloc((void **)&porganisms, size * sizeof(T*)), "Population porganisms hipMalloc");
    CUDA_CALL(hipMalloc((void **)&children, size * sizeof(T)), "Population children hipMalloc");
    CUDA_CALL(hipMalloc((void **)&pchildren, size * sizeof(T*)), "Population pchildren hipMalloc");
    CUDA_CALL(hipMalloc((void **)&ichildren, size * sizeof(bool)), "Population ichildren hipMalloc");
    InitKernel<<<size / 1024 + 1, 1024>>>(organisms, porganisms, children, pchildren, size);
    CUDA_CALL(hipStreamCreate(&stream), "Population stream create");
}

template<typename T> Population<T>::~Population(){
    CUDA_CALL(hipFree(organisms), "Population organisms hipFree");
    CUDA_CALL(hipFree(porganisms), "Population porganisms hipFree");
    CUDA_CALL(hipFree(children), "Population children hipFree");
    CUDA_CALL(hipFree(pchildren), "Population pchildren hipFree");
    CUDA_CALL(hipFree(ichildren), "Population ichildren hipFree");
    CUDA_CALL(hipStreamDestroy(stream), "Population stream destroy");
}

template<typename T> template<typename r> void Population<T>::random(r a, r b){
    RandomKernel<<<size / 1024 + 1, 1024, 0, stream>>>(organisms, time(NULL), size, a, b);
}

template<typename T> template<typename r> void Population<T>::brandom(r a, r b, int nthreads){
    BRandomKernel<<<size, nthreads, 0, stream>>>(organisms, time(NULL), a, b);
}

template<typename T> template<typename r> void Population<T>::linspace(r a, r b, bool endpoint){
    LinspaceKernel<<<size / 1024 + 1, 1024, 0, stream>>>(organisms, size, a, b, endpoint);
}

template<typename T> template<typename r> void Population<T>::blinspace(r a, r b, bool endpoint){
    BLinspaceKernel<<<size, 1024, 0, stream>>>(organisms, size, a, b, endpoint);
}

template<typename T> template<typename r> void Population<T>::plinspace(r a, r b, bool endpoint){
    PLinspaceKernel<<<size / 1024 + 1, 1024, 0, stream>>>(organisms, size, a, b, endpoint);
}

template<typename T> template<typename r> void Population<T>::bplinspace(r a, r b, bool endpoint){
    BPLinspaceKernel<<<size, 1024, 0, stream>>>(organisms, size, a, b, endpoint);
}

template<typename T> template<typename r> void Population<T>::logspace(r a, r b, DNA base, bool endpoint){
    LogspaceKernel<<<size / 1024 + 1, 1024, 0, stream>>>(organisms, size, a, b, base, endpoint);
}

template<typename T> template<typename r> void Population<T>::blogspace(r a, r b, DNA base, bool endpoint){
    BLogspaceKernel<<<size, 1024, 0, stream>>>(organisms, size, a, b, base, endpoint);
}

template<typename T> template<typename r> void Population<T>::plogspace(r a, r b, DNA base, bool endpoint){
    PLogspaceKernel<<<size / 1024 + 1, 1024, 0, stream>>>(organisms, size, a, b, base, endpoint);
}

template<typename T> template<typename r> void Population<T>::bplogspace(r a, r b, DNA base, bool endpoint){
    BPLogspaceKernel<<<size, 1024, 0, stream>>>(organisms, size, a, b, base, endpoint);
}

template<typename T> void Population<T>::fitness(){
    FitnessKernel<<<size / 1024 + 1, 1024, 0, stream>>>(organisms, size);
}

template<typename T> void Population<T>::bfitness(int nthreads){
    BFitnessKernel<T, Tfitness><<<size, nthreads, nthreads * sizeof(Tfitness), stream>>>(organisms, nthreads);
}

template<typename T> void Population<T>::mutate(MUTATION mutation_type, float probability){
    switch (mutation_type)
    {
        case MUTATION_INVERSION:
            MutationInversionKernel<<<size / 1024 + 1, 1024, 0, stream>>>(organisms, size, probability, time(NULL));
            break;
        case MUTATION_OWN:
            MutationOwnKernel<<<size / 1024 + 1, 1024, 0, stream>>>(organisms, size, probability, time(NULL));
            break;
        case MUTATION_SCRAMBLE:
            MutationScrambleKernel<<<size / 1024 + 1, 1024, 0, stream>>>(organisms, size, probability, time(NULL));        
            break;
        case MUTATION_SWAP:
            MutationSwapKernel<<<size / 1024 + 1, 1024, 0, stream>>>(organisms, size, probability, time(NULL));
            break;
    }
}

template<typename T> void Population<T>::crossover(CROSSOVER crossover_type, float probability){
    int children_size = size * probability / 100;
    switch (crossover_type)
    {
        case CROSSOVER_ARITHMETIC:
            CrossoverArithmeticKernel<<<children_size / 1024 + 1, 1024, 0, stream>>>(organisms, children, ichildren, children_size, time(NULL));
            break;
        case CROSSOVER_OWN:
            CrossoverOwnKernel<<<children_size / 1024 + 1, 1024, 0, stream>>>(organisms, children, ichildren, children_size, time(NULL));
            break;
        case CROSSOVER_SINGLE_POINT:
            CrossoverSinglePointKernel<<<children_size / 1024 + 1, 1024, 0, stream>>>(organisms, children, ichildren, children_size, time(NULL));
            break;
        case CROSSOVER_TWO_POINT:
            CrossoverTwoPointKernel<<<children_size / 1024 + 1, 1024, 0, stream>>>(organisms, children, ichildren, children_size, time(NULL));
            break;
        case CROSSOVER_UNIFORM:
            CrossoverUniformKernel<<<children_size / 1024 + 1, 1024, 0, stream>>>(organisms, children, ichildren, children_size, time(NULL));
            break;
    }
}

template<typename T> void Population<T>::sortAll(){
    for(int k = 2; k <= 2 * size; k <<= 1){
        for(int j = k >> 1; j > 0; j >>= 1){
            SortAllKernel<<<size / 1024 + 1, 1024, 0, stream>>>(porganisms, pchildren, ichildren, size, j, k);
        }
    }
}

template<typename T> void Population<T>::sortOrganisms(){
    for(int k = 2; k <= 2 * size; k <<= 1){
        for(int j = k >> 1; j > 0; j >>= 1){
            BitonicSortKernel<<<size / 1024 + 1, 1024, 0, stream>>>(porganisms, size, j, k);
        }
    }
}

template<typename T> void Population<T>::init_organisms(){
    InitOrganismsKernel<<<size / 1024 + 1, 1024, 0, stream>>>(organisms, size);
}

template<typename T> void Population<T>::init_organisms_with_tid(){
    InitOrganismsWithTidKernel<<<size / 1024 + 1, 1024, 0, stream>>>(organisms, size);
}

template<typename T> void Population<T>::init_organisms_with_val(DNA val){
    InitOrganismsWithValKernel<<<size / 1024 + 1, 1024, 0, stream>>>(organisms, size, val);
}

template<typename T> void Population<T>::binit_organisms_with_val(DNA val){
    BInitOrganismsWithValKernel<<<size, 1024, 0, stream>>>(organisms, size, val);
}

template<typename T> void Population<T>::printP(int max){
    PrintPointersKernel<<<1, 1, 0, stream>>>(porganisms, size, max == -1 ? size:max);
}

template<typename T> void Population<T>::print(int max){
    PrintKernel<<<1, 1, 0, stream>>>(organisms, size, max == -1 ? size:max);
}

template<typename T> void Population<T>::print_childrenP(int max){
    PrintChildrenKernelP<<<1, 1, 0, stream>>>(pchildren, ichildren, size, max == -1 ? size:max);
}

template<typename T> void Population<T>::print_children(int max){
    PrintChildrenKernel<<<1, 1, 0, stream>>>(children, ichildren, size, max == -1 ? size:max);
}

template<typename T> hipStream_t* Population<T>::getStream(){
    return &stream;
}

#endif // SPOPULATION_CU
