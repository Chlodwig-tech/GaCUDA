#ifndef TIMER_CU
#define TIMER_CU

#include "stdio.h"

class Timer{
    hipEvent_t estart, estop;
    hipStream_t *stream;
    float milliseconds;
public:
    Timer(hipStream_t *cstream=NULL){
        hipEventCreate(&estart);
        hipEventCreate(&estop);
        milliseconds = 0;
        stream = cstream;
    }
    void start(){
        if(stream != NULL)
            hipEventRecord(estart, *stream);
        else
            hipEventRecord(estart);
    }
    void stop(){
        if(stream != NULL)
            hipEventRecord(estop, *stream);
        else
            hipEventRecord(estop);
        hipEventSynchronize(estop);
        hipEventElapsedTime(&milliseconds, estart, estop);
    }
    float get(){
        return milliseconds;
    }
    void print(){
        printf("Time taken: %fms\n", milliseconds);
    }
};

#endif // TIMER_CU
