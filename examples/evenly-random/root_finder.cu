#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <fstream>
#include <sstream>
#include <iomanip>
#include <cstdlib>

#include "../../gacuda/gacuda.h"

template<int Size> class RootFinder : public Organism<float, float, Size>{
public:
    __device__ void print(){
        for(int i = 0; i < Size; i++){
            printf("%f ", this->genes[i]);
        }
        printf("-> %f\n", this->fvalue);
    }
    __device__ void fitness(){
        // f(x0, x1, x2, x3, x4, x5, x6, x7) =
        // = x0^7 + 3x1^6 - x2^5 + 2.4x3^4 - 1.02x4^3 - x5 +12x6^2 - x7
        float f = 0;
        f += pow(this->genes[0], 7);
        f += 3 * pow(this->genes[1], 6);
        f += pow(this->genes[2], 5);
        f += 2.4 * pow(this->genes[3], 4);
        f += 1.02 * pow(this->genes[4], 3);
        f += -this->genes[5];
        f += 12 * pow(this->genes[6], 2);
        f += -this->genes[7];

        this->fvalue = abs(f);
    }
};


int main(int argc, char *argv[]){
    float good_solution = 0.999f;
   

    const int individual_size = 8; // individual size
    const int population_size = std::atoi(argv[1]); // population size
    float mutation_probability = std::atoi(argv[2]);
    float crossover_probability = std::atoi(argv[3]);

    std::stringstream filename;
    filename << "results/root_finder/output-" << population_size << "-" << mutation_probability << "-" << crossover_probability << ".txt";
    std::ofstream outFile(filename.str());
    outFile << std::fixed << std::setprecision(6);

    
    Population<RootFinder<individual_size>> p(population_size);

    for(int ii = 0; ii < 5; ii++){
        int x = -1;
        p.plinspace(-10.0f, 10.0f);
        p.random(-10.0f, 10.0f, population_size / 2);
        // p.random(-10.0f, 10.0f);
        p.set_current_best(9999.0f);
        p.fitness();
        //p.printP(2);
        int number_of_epochs = 50000;
        for(int i = 0; i < number_of_epochs; i++){
            if(i % 5000 == 0){
                float best = p.get_best_value();
                outFile << best << " ";
                if(x == -1 && best <= good_solution){
                    x = i;
                }
            }
            p.shift_mutate(0.001f, mutation_probability);
            //p.mutate(MUTATION_SCRAMBLE, mutation_probability);
            p.crossover(CROSSOVER_UNIFORM, crossover_probability); // 20% of the best organisms
            p.sortAll(); // sort all organisms and make selection
        }
        float best = p.get_best_value();
        if(x == -1 && best <= good_solution){
            x = number_of_epochs;
        }
        outFile << best << " | " << x << "\n";
        p.reset_children();
    }
    outFile.close();

    //p.printP(2);
    //p.print_current_best();
    hipDeviceSynchronize();
}